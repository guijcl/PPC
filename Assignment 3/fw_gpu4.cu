#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "workshop.h"

#define GRAPH_SIZE 2048
//#define GRAPH_SIZE 1024
//#define GRAPH_SIZE 512

#define THREADS_PER_BLOCK 32
#define BLOCKS GRAPH_SIZE/THREADS_PER_BLOCK

#define EDGE_COST(graph, graph_size, a, b) graph[a * graph_size + b]
#define D(a, b) EDGE_COST(output, graph_size, a, b)

#define INF 0x1fffffff

/*void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        int r;
        r = rand() % 40 + 6;
        if (r > 20) {
          r = INF;
        }

        D(i, j) = r;
      }
    }
  }
}*/

//NEW ARRAY GEN
void generate_random_graph(int *output, int graph_size) {
  int i, j;

  srand(0xdadadada);

  for (i = 0; i < graph_size; i++) {
    for (j = 0; j < graph_size; j++) {
      if (i == j) {
        D(i, j) = 0;
      } else {
        D(i, j) = i * j;
        if (i % 2==0)
          D(i, j) += j * 4;
      }
    }
  }
}

__global__ void floyd_warshall_kernel(int graph_size, int *output) {
    //SLOW - SHARED MEMORY NOT WORKING AND OVERALL LOGIC IS WRONG
    int i = blockIdx.x * blockDim.x + threadIdx.x; //col (THREADS_PER_BLOCK)
    int j = blockIdx.y * blockDim.y + threadIdx.y; //row (THREADS_PER_BLOCK)

    int index = threadIdx.x; //equivalent to 'i' but in cache (0 to THREADS_PER_BLOCK - 1)

    __shared__ int *cache; //cache = [THREADS_PER_BLOCK][GRAPH_SIZE]
    cache = (int*)malloc(THREADS_PER_BLOCK * GRAPH_SIZE * sizeof(int));
    __syncthreads();
    cache[index * THREADS_PER_BLOCK + j] = output[i * graph_size + j];
    __syncthreads();

    //printf("i:%d j:%d res:%d\n", i, j, cache[index * THREADS_PER_BLOCK + j]);

    for(int k = 0; k < graph_size; k++) {
        while(j < graph_size) {
        if(cache[index * THREADS_PER_BLOCK + k] + cache[k * THREADS_PER_BLOCK + j] < cache[index * THREADS_PER_BLOCK + j])
            atomicExch(&cache[index * THREADS_PER_BLOCK + j], cache[index * THREADS_PER_BLOCK + k] + cache[k * THREADS_PER_BLOCK + j]);
        j += blockDim.y * gridDim.y;
        __syncthreads();
        }
    }
    
    int j_tmp = blockIdx.y * blockDim.y + threadIdx.y;
    while(j_tmp < graph_size) {
        D(i, j_tmp) = cache[index * THREADS_PER_BLOCK + j_tmp];
        j_tmp += blockDim.y * gridDim.y;
    }
  
}

void floyd_warshall_gpu(const int *graph, int graph_size, int *output) {
  int *dev_output_gpu;

  int size = sizeof(int) * graph_size * graph_size;

  HANDLE_ERROR(hipMalloc(&dev_output_gpu, size));
  HANDLE_ERROR(hipMemcpy(dev_output_gpu, graph, size, hipMemcpyHostToDevice));

  for(int k = 0; k < graph_size; k++) {
    floyd_warshall_kernel<<<BLOCKS, THREADS_PER_BLOCK>>>(graph_size, dev_output_gpu, k);
  }

  printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));

  HANDLE_ERROR(hipMemcpy(output, dev_output_gpu, size, hipMemcpyDeviceToHost));
  hipFree(dev_output_gpu);
}

void floyd_warshall_cpu(const int *graph, int graph_size, int *output) {
  int i, j, k;

  memcpy(output, graph, sizeof(int) * graph_size * graph_size);

  for (k = 0; k < graph_size; k++) {
    for (i = 0; i < graph_size; i++) {
      for (j = 0; j < graph_size; j++) {
        if (D(i, k) + D(k, j) < D(i, j)) {
          D(i, j) = D(i, k) + D(k, j);
        }
      }
    }
  }
}

int main(int argc, char **argv) {
#define TIMER_START() gettimeofday(&tv1, NULL)
#define TIMER_STOP()                                                           \
  gettimeofday(&tv2, NULL);                                                    \
  timersub(&tv2, &tv1, &tv);                                                   \
  time_delta = (float)tv.tv_sec + tv.tv_usec / 1000000.0

  struct timeval tv1, tv2, tv;
  float time_delta;

  int *graph, *output_cpu, *output_gpu;

  int size;
  size = sizeof(int) * GRAPH_SIZE * GRAPH_SIZE;

  graph = (int *)malloc(size);
  assert(graph);

  output_cpu = (int *)malloc(size);
  assert(output_cpu);
  memset(output_cpu, 0, size);

  output_gpu = (int *)malloc(size);
  assert(output_gpu);

  generate_random_graph(graph, GRAPH_SIZE);

  printf("BLOCKS: %d\nTHREADS PER BLOCK: %d\n\n", BLOCKS, THREADS_PER_BLOCK);

  fprintf(stderr, "running on cpu...\n");
  TIMER_START();
  floyd_warshall_cpu(graph, GRAPH_SIZE, output_cpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  fprintf(stderr, "running on gpu...\n");
  TIMER_START();
  floyd_warshall_gpu(graph, GRAPH_SIZE, output_gpu);
  TIMER_STOP();
  fprintf(stderr, "%f secs\n", time_delta);

  //if (memcmp(output_cpu, output_gpu, size) != 0)
  //  fprintf(stderr, "FAIL!\n");

  for(int i = 0; i < GRAPH_SIZE; i++) {
    if(output_cpu[i] != output_gpu[i]) {
      printf("FAIL: Values do not match..\n");
      printf("Index: %d   CPU: %d   GPU %d\n", 0, output_cpu[0], output_gpu[0]);
      printf("Index: %d   CPU: %d   GPU %d\n", i, output_cpu[i], output_gpu[i]);
      break;
    }
  }
  //printf("SUCCESSFUL: It's correct!\n");

  return 0;
}
